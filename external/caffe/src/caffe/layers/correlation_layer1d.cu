#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/output.hpp"

#include "caffe/layers/correlation_1d_layer.hpp"

#include "caffe/caffe.hpp"

#define ROUND_OFF 50000

#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

namespace caffe {

// == Dimension rearrangement Kernel
  
namespace corr1d {
  
template <typename Dtype>
__global__ void blob_rearrange_kernel2(const Dtype* in, Dtype* out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    int xy = blockIdx.x*blockDim.x + threadIdx.x;
    if(xy>=widthheight)
        return;

    int ch = blockIdx.y;
    int n  = blockIdx.z;

    float value=in[(n*channels+ch)*widthheight+xy];

    __syncthreads();

    int xpad  = (xy % width + padding);
    int ypad  = (xy / width + 0);
    int xypad = ypad * (width+2*padding) + xpad;

    out[(n*pwidthheight+xypad)*channels + ch] = value;
}

// == Correlation Kernel
template <typename Dtype> 
__global__ void CorrelateData(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) 
{
  extern __shared__ char patch_data_char[];
  
  Dtype *patch_data = (Dtype *)patch_data_char;
  
    // First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
  int x1 = blockIdx.x*stride1 + max_displacement;
  int y1 = blockIdx.y*stride1;
  int item = blockIdx.z;
  int ch_off = threadIdx.x;
  
  // Load 3D patch into shared shared memory
  for(int j = 0; j < kernel_size; j++) { // HEIGHT
    for(int i = 0; i < kernel_size; i++) { // WIDTH
      int ji_off = ((j * kernel_size) + i) * bottomchannels;
      for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
          int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
          int idxPatchData = ji_off + ch;
          patch_data[idxPatchData] = bottom0[idx1];
      }
    }
  }
  
  __syncthreads();
  
  __shared__ Dtype sum[WARPS_PER_BLOCK*THREADS_PER_WARP];
  
  // Compute correlation
  for(int top_channel = 0; top_channel < topchannels; top_channel++) {
    sum[ch_off] = 0;
  
    int s2o = (top_channel % neighborhood_grid_width + x_shift) * stride2;
    
    for(int j = 0; j < kernel_size; j++) { // HEIGHT
      for(int i = 0; i < kernel_size; i++) { // WIDTH
        int ji_off = ((j * kernel_size) + i) * bottomchannels;
        for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
          int x2 = x1 + s2o;
          
          int idxPatchData = ji_off + ch;
          int idx2 = ((item * bottomheight + y1+j) * bottomwidth + x2+i) * bottomchannels + ch;
          
          sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
        }
      }
    }
    
    __syncthreads();
    
    if(ch_off == 0) {
        Dtype total_sum = 0;
        for(int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
            total_sum += sum[idx];
        }
        const int sumelems = kernel_size*kernel_size*bottomchannels;
        const int index = ((top_channel*topheight + blockIdx.y)*topwidth)+blockIdx.x;
        top[index + item*topcount] = total_sum / (float)sumelems;
    }
  }
  
  
  // Aggregate  
}

// == Correlation Backward Pass Kernel (For Blob 0)
template <typename Dtype> 
__global__ void CorrelateDataBackward0(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom1, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight; //h-pos

    //Get X,Y ranges and clamp
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    
    // Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
    int ymax = (m - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1
    

    Dtype sum = 0;
    if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
    {
        xmin = max(0,xmin);
        xmax = min(topwidth-1,xmax);

        ymin = max(0,ymin);
        ymax = min(topheight-1,ymax);

        {
          for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

            // Get bottom1 data:
            int s2o = stride2 * o;
            int idxbot1 = ((item * pbottomheight + m) * pbottomwidth + (l+s2o)) * bottomchannels + n;
            Dtype bot1tmp = bottom1[idxbot1]; // bottom1[l+s2o,m,n]

            // Index offset for topdiff in following loops:
            int op = (o-x_shift); // index [o,p]
            int idxopoffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot1tmp;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
		const int bot0index = ((n * bottomheight) + m) * bottomwidth + (l-pad_size);
    bottom0diff[bot0index + item*bottomcount] = sum / (float)sumelems;
  }

}



// == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype> 
__global__ void CorrelateDataBackward1(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, Dtype *bottom1diff, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    //int l = index % bottomwidth + pad_size; //w-pos
    //int m = (index / bottomwidth) % bottomheight + pad_size; //h-pos
    //int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight; //h-pos
    
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    Dtype sum = 0;
    {
      
      for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
        
        int s2o = stride2 * o;
        
        //Get X,Y ranges and clamp
        // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - 0 - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        
        // Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
        int ymax = (m - 0 - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - 0) / stride1

        if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
        {
            xmin = max(0,xmin);
            xmax = min(topwidth-1,xmax);

            ymin = max(0,ymin);
            ymax = min(topheight-1,ymax);

            // Get bottom0 data:
            int idxbot0 = ((item * pbottomheight + m) * pbottomwidth + (l-s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot0]; // bottom1[l+s2o,m,n]

            // Index offset for topdiff in following loops:
            int op = (o-x_shift); // index [o,p]
            int idxOpOffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot0tmp;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
		const int bot1index = ((n * bottomheight) + m) * bottomwidth + (l-pad_size);
		bottom1diff[bot1index + item*bottomcount] = sum / (float)sumelems;
  }

}

// == Correlation Kernel Subtraction
template <typename Dtype> 
__global__ void CorrelateDataSubtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int x = index % topwidth; //w-pos
    int y = (index / topwidth) % topheight; //h-pos
    int c = (index / topwidth / topheight) % topchannels; //channels
        
    // Offset of patch in image 2
    int s2o = (c % neighborhood_grid_width + x_shift) * stride2;
        
    // First (upper left) position of kernel center in current neighborhood in image 1
    int x1 = x*stride1 + kernel_radius + max_displacement;
    int y1 = y*stride1 + kernel_radius + 0;
    
    // Iterate through 3D patch
    Dtype sum = 0;
    for(int j = -kernel_radius; j <= kernel_radius; j++) { // HEIGHT
      for(int i = -kernel_radius; i <= kernel_radius; i++) { // WIDTH
        for(int l = 0; l < bottomchannels; l++) { // CHANNELS
          // Calculate position in image 2
          int x2 = x1 + s2o;
          int y2 = y1;

          // Indices in bottom data: (CH=l,W=x2,H=y2,N)
          int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + l;
          int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + l;

          // Do the correlation:
          sum += fabsf(bottom0[idx1] - bottom1[idx2]);
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    top[index + item*topcount] = sum / (float)sumelems;
  }

}


// == Correlation Backward Pass Kernel (For Blob 0)
template <typename Dtype> 
__global__ void CorrelateDataBackward0Subtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom0, const Dtype *bottom1, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int l = index % bottomwidth + pad_size; //w-pos
    int m = (index / bottomwidth) % bottomheight; //h-pos
    int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels

    //Get X,Y ranges and clamp
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    
    // Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
    int ymax = (m - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1
    

    Dtype sum = 0;
    if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
    {
        xmin = max(0,xmin);
        xmax = min(topwidth-1,xmax);

        ymin = max(0,ymin);
        ymax = min(topheight-1,ymax);

        { 
          for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

            // Get bottom1 data:
            int s2o = stride2 * o;
            int idxbot = ((item * pbottomheight + (m)) * pbottomwidth + (l+s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m,n]
            Dtype bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m,n]
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(1.0) : Dtype(-1.0);

            // Index offset for topdiff in following loops:
            int op = (o-x_shift); // index [o,p]
            int idxopoffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    bottom0diff[index + item*bottomcount] = sum / (float)sumelems;
  }

}


// == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype> 
__global__ void CorrelateDataBackward1Subtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *bottom1diff, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int l = index % bottomwidth + pad_size; //w-pos
    int m = (index / bottomwidth) % bottomheight; //h-pos
    int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels
    
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    Dtype sum = 0;
    {
      for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
        
        int s2o = stride2 * o;
        
        //Get X,Y ranges and clamp
        // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - 0 - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        
        // Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
        int ymax = (m - 0 - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - s2p) / stride1

        if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
        {
            xmin = max(0,xmin);
            xmax = min(topwidth-1,xmax);

            ymin = max(0,ymin);
            ymax = min(topheight-1,ymax);

            // Get bottom0 data:
            int idxbot = ((item * pbottomheight + (m)) * pbottomwidth + (l-s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m,n]
            Dtype bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m,n]
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(-1.0) : Dtype(1.0);

            // Index offset for topdiff in following loops:
            int op = (o-x_shift); // index [o,p]
            int idxOpOffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    bottom1diff[index + item*bottomcount] = sum / (float)sumelems;
  }

}

} // end namespace

// == Forward 

template <typename Dtype>
void Correlation1DLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top)
{
    CHECK_EQ(bottom.size(),2);
    CHECK_EQ(top.size(),1);

    const int bnum = bottom[0]->num();
    const int bchannels = bottom[0]->channels();
    const int bheight = bottom[0]->height();
    const int bwidth = bottom[0]->width();
    const int bwidthheight = bwidth * bheight;

    const int topcount = top_width_ * top_height_ * top_channels_;
    
    dim3 threadsPerBlock(THREADS_PER_WARP * WARPS_PER_BLOCK); 
    
    hipMemset(rbot1_->mutable_gpu_data(), 0, rbot1_->count()*sizeof(Dtype));
    hipMemset(rbot2_->mutable_gpu_data(), 0, rbot2_->count()*sizeof(Dtype));
    
    int threads_per_block=16;
    dim3 totalBlocksRearr((bwidthheight-1)/threads_per_block+1, bchannels, bnum);
    const int pwidthheight = (bwidth + 2 * pad_size_) * (bheight);
    
    corr1d::blob_rearrange_kernel2<Dtype><<<totalBlocksRearr,threads_per_block>>>
            (bottom[0]->gpu_data(),rbot1_->mutable_gpu_data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);
    
    corr1d::blob_rearrange_kernel2<Dtype><<<totalBlocksRearr,threads_per_block>>>
            (bottom[1]->gpu_data(),rbot2_->mutable_gpu_data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);
    
    const int num = bnum;
    const int channels = bchannels;
    const int height = bheight;
    const int width = bwidth + 2*pad_size_;
    
    const int shared_memory_per_block = (kernel_size_*kernel_size_)*bchannels;

    int x_shift = - neighborhood_grid_radius_;
    if(single_direction_ == -1) { // to the left
      x_shift = -neighborhood_grid_width_;
    } else if(single_direction_ == 1) { // to the right
      x_shift = 0;
    }
    
    if(corr_type_ == CorrelationParameter_CorrelationType_MULTIPLY) {
        // Correlation1DLayer
        int topThreadCount = topcount;
        
        dim3 totalBlocksCorr(top_width_, top_height_, num);
        
        
        corr1d::CorrelateData<Dtype><<<totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(Dtype)>>>(
            topThreadCount,
            num, top_width_, top_height_, top_channels_, topcount,
            max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_, kernel_size_,
            stride1_, stride2_,
            width, height, channels,
            rbot1_->gpu_data(), rbot2_->gpu_data(), top[0]->mutable_gpu_data()
            );

        CUDA_POST_KERNEL_CHECK;
        
    } else if(corr_type_ == CorrelationParameter_CorrelationType_SUBTRACT) {
        // Correlation1DLayer
        for(int n = 0; n < num; n++) {
            
            int topThreadCount = topcount;
            
            corr1d::CorrelateDataSubtract<Dtype><<<CAFFE_GET_BLOCKS(topThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
                topThreadCount,
                num, n, top_width_, top_height_, top_channels_, topcount,
                max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_,
                stride1_, stride2_,
                width, height, channels,
                rbot1_->gpu_data(), rbot2_->gpu_data(), top[0]->mutable_gpu_data()
                );

            
            CUDA_POST_KERNEL_CHECK;
        }
    }
}


template <typename Dtype>
void Correlation1DLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{

    // Get top diff, compute bottom diff
    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom0_diff = bottom[0]->mutable_gpu_diff();
    Dtype* bottom1_diff = bottom[1]->mutable_gpu_diff();

    const Dtype* bottom0_data = bottom[0]->gpu_data();
    const Dtype* bottom1_data = bottom[1]->gpu_data();

    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();

    const int paddedheight = height;
    const int paddedwidth = width + 2*pad_size_;

    const int bottomcount = channels * height * width;

    int botThreadCount = bottomcount;
   
    // CorrelationLayerBackward
    
    bottom0_diff = bottom[0]->mutable_gpu_diff();
    bottom1_diff = bottom[1]->mutable_gpu_diff();
    
    int x_shift = - neighborhood_grid_radius_;
    if(single_direction_ == -1) { // to the left
      x_shift = -neighborhood_grid_width_;
    } else if(single_direction_ == 1) { // to the right
      x_shift = 0;
    }
    
    if(corr_type_ == CorrelationParameter_CorrelationType_MULTIPLY) {
        
        // == Run kernel Backward 0
        dim3 totalBlocksBackward0(width, height, channels * num); //First dim is fastest
        dim3 threadsPerBlockBackward0(THREADS_PER_WARP * WARPS_PER_BLOCK); 
        const int buffer_size_backw0 = ((int)ceil((float)(2 * kernel_radius_) / (float)stride1_) + 1) * top_channels_;
       
        // == Run kernel Backward 0 
        for(int n = 0; n < num; n++) {
        //Bottom0:
        corr1d::CorrelateDataBackward0<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot2_->gpu_data(), top_diff
            ); 
    
        CUDA_POST_KERNEL_CHECK;
        }
        
        // == Run kernel Backward 1
        for(int n = 0; n < num; n++) {
        corr1d::CorrelateDataBackward1<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1_->gpu_data(), bottom1_diff, top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }
        
    } else if(corr_type_ == CorrelationParameter_CorrelationType_SUBTRACT) {
        for(int n = 0; n < num; n++) {
        //Bottom0:
        corr1d::CorrelateDataBackward0Subtract<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot1_->gpu_data(), rbot2_->gpu_data(), top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }

        for(int n = 0; n < num; n++) {
        //Bottom0:
        corr1d::CorrelateDataBackward1Subtract<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, x_shift, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1_->gpu_data(), rbot2_->gpu_data(), bottom1_diff, top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }
    }
}


INSTANTIATE_LAYER_GPU_FUNCS(Correlation1DLayer);

}  // namespace caffe
