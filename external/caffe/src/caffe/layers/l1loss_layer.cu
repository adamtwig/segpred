#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/l1_loss_layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ComputeSign(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? Dtype(1) : Dtype(-1);
  }
} 

// TODO maybe change the way of detecting NaNs

template <typename Dtype>
__global__ void FindNotNaNs(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index]==in[index] ? Dtype(1) : Dtype(0);
  }
} 

template <typename Dtype>
__global__ void KillNaNs(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index]==in[index] ? in[index] : Dtype(0);
  }
}

template <typename Dtype>
__global__ void KillMasked(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > Dtype(0.5) ? out[index] : Dtype(0);
//     out[index] = out[index]==out[index] ? out[index] : Dtype(0);
//     out[index] = out[index]>1e3 ? 0 : out[index];
//     out[index] = out[index]<-1e3 ? 0 : out[index];
  }
}

template <typename Dtype>
__global__ void KillMaskedAcrossChannels(const int n, const int width_height, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int mask_idx = index % width_height;
    out[index] = in[mask_idx] > Dtype(0.5) ? out[index] : Dtype(0);
  }
}

template <typename Dtype>
__global__ void MaskPlateauValues(const int n, const Dtype* in, Dtype* out, Dtype plateau) {
  CUDA_KERNEL_LOOP(index, n) {
    if(fabs(in[index]) < plateau) out[index] = Dtype(0); // Mask out plateau values and keep other as is
  }
} 

template <typename Dtype>
__global__ void MaskPlateauValuesInitial(const int n, const Dtype* in, Dtype* out, Dtype plateau) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = (fabs(in[index]) < plateau) ? Dtype(0) : Dtype(1);
  }
} 


template <typename Dtype>
void L1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top)
{
  
  Blob<Dtype> *diffptr = diff_top_vec_[0];
  
  Dtype dot, loss;
  if(bottom.size() > 1) {
    diff_layer_->Forward(bottom, diff_top_vec_);
  }
  
  // if necessary, compute the number of not-NaNs
  int count = bottom[0]->count();
  int num = bottom[0]->num();
  FindNotNaNs<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, diffptr->gpu_data(), mask_.mutable_gpu_data());
  hipDeviceSynchronize();
  CUDA_POST_KERNEL_CHECK;
  
  if (this->layer_param_.l1_loss_param().normalize_by_num_entries()) {    
    caffe_gpu_dot(count, mask_.gpu_data(), mask_.gpu_data(), &normalize_coeff_);
    normalize_coeff_ /= mask_.channels();
  } else {
    normalize_coeff_ = num;
  }
  
  if (this->layer_param_.l1_loss_param().l2_per_location()) {
    // set masked (NaNs only) to zero
    KillMasked<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, mask_.gpu_data(), diffptr->mutable_gpu_data());
    hipDeviceSynchronize();
    CUDA_POST_KERNEL_CHECK;
    
    square_layer_->Forward(diff_top_vec_, square_top_vec_);
    sum_layer_->Forward(square_top_vec_, sum_top_vec_);
    
    // Mask plateau in summed blob (only one channel):
    if(this->layer_param_.l1_loss_param().plateau() > 0) {
      float plateau_val_squared = this->layer_param_.l1_loss_param().plateau() * this->layer_param_.l1_loss_param().plateau();
      MaskPlateauValuesInitial<Dtype><<<CAFFE_GET_BLOCKS(sum_output_.count()), CAFFE_CUDA_NUM_THREADS>>>(
          sum_output_.count(), sum_output_.gpu_data(), plateau_l2_.mutable_gpu_data(), plateau_val_squared);
      hipDeviceSynchronize();
      CUDA_POST_KERNEL_CHECK;
      
      KillMasked<Dtype><<<CAFFE_GET_BLOCKS(sum_output_.count()), CAFFE_CUDA_NUM_THREADS>>>(
            sum_output_.count(), plateau_l2_.gpu_data(), sum_output_.mutable_gpu_data());
      hipDeviceSynchronize();
      CUDA_POST_KERNEL_CHECK;
    }
    
    sqrt_layer_->Forward(sum_top_vec_, sqrt_top_vec_);
    // Note sign_ is set to all ones in Reshape
    caffe_gpu_dot(sqrt_output_.count(), sqrt_output_.gpu_data(), sign_.gpu_data(), &dot);
  }
  else {    
    // Mask plateau:
    if(this->layer_param_.l1_loss_param().plateau() > 0) {
      MaskPlateauValues<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, diffptr->gpu_data(), mask_.mutable_gpu_data(), this->layer_param_.l1_loss_param().plateau());
      CUDA_POST_KERNEL_CHECK;
    }
    
    //mask_.print("MASK2");
    
    // set masked (NaNs, plateau) to zero
    KillMasked<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, mask_.gpu_data(), diffptr->mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;
    
    ComputeSign<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, diffptr->gpu_data(), sign_.mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;
    caffe_gpu_dot(count, diffptr->gpu_data(), sign_.gpu_data(), &dot); 
  }
  loss = dot / normalize_coeff_; 
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void L1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{  
  bool prop_down = propagate_down[0];
  if(bottom.size() > 1) prop_down |= propagate_down[1];
  
  Blob<Dtype> *diffptr = diff_top_vec_[0];
  
  if (prop_down) {
    const Dtype alpha = top[0]->cpu_diff()[0] / normalize_coeff_;
    if (this->layer_param_.l1_loss_param().l2_per_location()) {
      vector<bool> prop_down(1,true);
      caffe_gpu_axpby(sqrt_output_.count(), alpha, sign_.gpu_data(),                   
          Dtype(0), sqrt_output_.mutable_gpu_diff());
      sqrt_layer_->Backward(sqrt_top_vec_, prop_down, sum_top_vec_);
      
      if(this->layer_param_.l1_loss_param().plateau() > 0) {
        KillMasked<Dtype><<<CAFFE_GET_BLOCKS(sum_output_.count()), CAFFE_CUDA_NUM_THREADS>>>(
              sum_output_.count(), plateau_l2_.gpu_data(), sum_output_.mutable_gpu_diff());
        hipDeviceSynchronize();
        CUDA_POST_KERNEL_CHECK;
      }
      
      sum_layer_->Backward(sum_top_vec_, prop_down, square_top_vec_);
      square_layer_->Backward(square_top_vec_, prop_down, diff_top_vec_);
      
    
    }
    else {    
      caffe_gpu_axpby(diffptr->count(), alpha, sign_.gpu_data(), 
          Dtype(0), diffptr->mutable_gpu_diff());
    }
    
    KillMasked<Dtype><<<CAFFE_GET_BLOCKS(diffptr->count()), CAFFE_CUDA_NUM_THREADS>>>(
        diffptr->count(), mask_.gpu_data(), diffptr->mutable_gpu_diff());
    CUDA_POST_KERNEL_CHECK;
    
    if(bottom.size() > 1) {
        diff_layer_->Backward(diff_top_vec_, propagate_down, bottom);
    }
  }
  
}

INSTANTIATE_LAYER_GPU_FUNCS(L1LossLayer);

}  // namespace caffe
